#define VIENNACL_WITH_CUDA
#include <vector>
#include <map>
#include <set>
#include <cstdlib>
#include <cmath>
#include <armadillo>
#include <iostream>
#include <fstream>
#include <iterator>
#include <algorithm>
#include <sstream>
#include <viennacl/matrix.hpp>
#include <viennacl/scalar.hpp>
#include <viennacl/vector.hpp>
#include <viennacl/fft.hpp>
#include <viennacl/linalg/fft_operations.hpp>

using namespace std;
using namespace arma;

namespace vcl = viennacl;

int main()
{
	vcl::matrix<float> mat_a(1000, 1000);
	vcl::matrix<float> mat_b(1000, 1000);

	while (true)
	{
		vcl::fft(mat_a, mat_b);
		cout << 'a';
	}

	return 0;
}

